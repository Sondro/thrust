#include <thrust/device_vector.h>
#include <thrust/functional.h>

#include <cub/device/device_scan.cuh>

void unused_gpu()
{
  // Invoke the same signature of hipcub::DeviceScan::ExclusiveScan as in bar.cu.
  // This function is never executed, it only has to be defined in libfoo.
  const std::size_t n{1024};
  thrust::device_vector<int> data(n);
  std::size_t                lvalue{};
  hipcub::DeviceScan::ExclusiveScan(nullptr,
                                 lvalue,
                                 data.begin(),
                                 data.begin(),
                                 thrust::plus<int>{},
                                 int{},
                                 int{});
}

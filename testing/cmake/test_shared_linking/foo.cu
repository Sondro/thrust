#include <thrust/device_vector.h>
#include <thrust/functional.h>

#include <cub/device/device_scan.cuh>

void unused()
{
  // Invoke the same signature of hipcub::DeviceScan::ExclusiveScan as in bar.cu.
  // This function is never executed, it only has to be defined in libfoo.
  const auto n = static_cast<std::size_t>(1024);
  thrust::device_vector<int> data(n);
  std::size_t                lvalue{};
  hipcub::DeviceScan::ExclusiveScan(nullptr,
                                 lvalue,
                                 data.begin(),
                                 data.begin(),
                                 thrust::plus<int>{},
                                 int{},
                                 int{});
}

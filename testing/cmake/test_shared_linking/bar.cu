#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/scan.h>
#include <thrust/sequence.h>

#include <hip/hip_runtime_api.h>

#include <cstdio>
#include <cstdlib>

int run_gpu(std::size_t size)
{
  thrust::device_vector<int> data(size);
  thrust::sequence(data.begin(), data.end());
  thrust::exclusive_scan(data.cbegin(), data.cend(), data.begin());
  const auto result = thrust::reduce(data.cbegin(), data.cend());
  printf("%s %d\n", "run_gpu:", result);

  const auto errCheck = hipGetLastError();
  printf("%s: [CUDA Error Check] %s\n",
         "run_gpu",
         hipGetErrorString(errCheck));
  if (errCheck != hipSuccess)
  {
    std::abort();
  }

  return result;
}
